#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <algorithm>      // std::random_shuffle (C++11)
#include <memory>         // std::auto_ptr (eliminado en C++17)
#include <type_traits>    // std::result_of (eliminado en C++20 / reemplazado)
#include <vector>
#include <numeric>
#include <hiprand/hiprand_kernel.h>
#include "sph_utils.hpp"

// Kernel simple: integra velocidad y posición con fuerza externa periódica
__global__ void integrate(float2* pos, float2* vel, int n, float dt, float t0, ForceParams fp){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=n) return;
  float t = t0;
  #pragma unroll 4
  for(int k=0;k<4;k++){
    float2 f = make_fext(t, fp);
    vel[i].x += f.x * dt;
    vel[i].y += f.y * dt;
    pos[i].x += vel[i].x * dt;
    pos[i].y += vel[i].y * dt;
    t += dt;
  }
}

// Utilidad host para “provocar” dependencia de estándar
struct Scale { float s; float operator()(float x) const { return s*x; } };

int main(){
  const int    N   = NPART;
  const int    STEPS_ = STEPS;
  const float  dt  = 5e-4f;

  // --- (1) Uso intencional de std::auto_ptr (DEPRECADO en C++11, ELIMINADO en C++17)
  std::auto_ptr<float> legacy_ptr(new float(1.0f)); // <- al portar a HIP con C++17: error

  // --- (2) Uso de std::result_of (OBSOLETO / reemplazar por std::invoke_result_t en C++17)
  typedef std::result_of<Scale(float)>::type result_t; // <- fallará en toolchains modernos
  (void)sizeof(result_t);

  // --- (3) Uso de std::random_shuffle (ELIMINADO en C++17; reemplazo -> std::shuffle)
  std::vector<int> ids(N);
  std::iota(ids.begin(), ids.end(), 0);
  std::random_shuffle(ids.begin(), ids.end()); // <- al portar a HIP: error de símbolo eliminado

  // Memoria host
  std::vector<float2> hpos(N), hvel(N);
  for(int i=0;i<N;i++){ hpos[i] = make_float2(0.f, 0.f); hvel[i] = make_float2(0.f, 0.f); }

  // Memoria device
  float2 *dpos=nullptr, *dvel=nullptr;
  hipMalloc(&dpos, N*sizeof(float2));
  hipMalloc(&dvel, N*sizeof(float2));
  hipMemcpy(dpos, hpos.data(), N*sizeof(float2), hipMemcpyHostToDevice);
  hipMemcpy(dvel, hvel.data(), N*sizeof(float2), hipMemcpyHostToDevice);

  const dim3 BS(BLOCK_SIZE);
  const dim3 GS((N + BS.x - 1)/BS.x);

  ForceParams fp; fp.fx0 = 0.0f; fp.fy0 = 0.0f; fp.amp = 2.0f; fp.w = 2.0f*3.1415926535f;

  hipEvent_t e0,e1;
  hipEventCreate(&e0); hipEventCreate(&e1);
  hipDeviceSynchronize();
  hipEventRecord(e0);

  float t = 0.f;
  for(int s=0;s<STEPS_;++s){
    integrate<<<GS,BS>>>(dpos, dvel, N, dt, t, fp);
    t += 4*dt;
  }

  hipEventRecord(e1);
  hipEventSynchronize(e1);
  float ms=0.f; hipEventElapsedTime(&ms, e0, e1);

  hipMemcpy(hpos.data(), dpos, N*sizeof(float2), hipMemcpyDeviceToHost);
  hipMemcpy(hvel.data(), dvel, N*sizeof(float2), hipMemcpyDeviceToHost);

  hipFree(dpos); hipFree(dvel);
  hipEventDestroy(e0); hipEventDestroy(e1);

  const double mpups = (double(N)*double(STEPS_)) / (ms/1000.0) * 1e-6;
  std::printf("B1 ExternalForces CUDA  N=%d steps=%d  Time=%.3f ms  MPUPS=%.3f  (block=%d)\n",
              N, STEPS_, ms, mpups, BLOCK_SIZE);
  return 0;
}
