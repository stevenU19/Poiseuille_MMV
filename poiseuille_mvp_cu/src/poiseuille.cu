#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>
#include <cmath>
#include "params.hpp"
#include "timer.hpp"
#include "sph_kernels.hpp"

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256
#endif
#ifndef YBINS
#define YBINS 64
#endif
#ifndef WRITE_PROFILE
#define WRITE_PROFILE 1
#endif

// SoA
struct Particles {
  float *x, *y, *vx, *vy, *ax, *ay, *rho, *P;
  int   *cellHead, *next;
  int    N, nxCells, nyCells;
  float  cellSize, h, rho0, cs, nu, gx, gy, Lx, Ly, dt, dp;
};

// --- Linked list ---
__global__ void kResetHeads(int *head, int n){
  int i=blockIdx.x*blockDim.x+threadIdx.x; if(i<n) head[i]=-1;
}
__global__ void kBuildLinked(const float* x,const float* y,int* next,int* head,
                             int N,float cellSize,int nx,int ny){
  int i=blockIdx.x*blockDim.x+threadIdx.x; if(i>=N) return;
  int cx = int(floorf(x[i]/cellSize)); cx = clampi(cx,0,nx-1);
  int cy = int(floorf(y[i]/cellSize)); cy = clampi(cy,0,ny-1);
  int c  = cellId(cx,cy,nx);
  next[i] = atomicExch(&head[c], i);
}

// --- Densidad y presión ---
__global__ void kDensityPressure(Particles p){
  int i=blockIdx.x*blockDim.x+threadIdx.x; if(i>=p.N) return;

  float xi=p.x[i], yi=p.y[i];
  int cx=int(floorf(xi/p.cellSize)); cx=clampi(cx,0,p.nxCells-1);
  int cy=int(floorf(yi/p.cellSize)); cy=clampi(cy,0,p.nyCells-1);

  float rhoi = 0.0f;
  #pragma unroll
  for(int oy=-1; oy<=1; ++oy){
    #pragma unroll
    for(int ox=-1; ox<=1; ++ox){
      int ncx = clampi(cx+ox,0,p.nxCells-1);
      int ncy = clampi(cy+oy,0,p.nyCells-1);
      int c   = cellId(ncx,ncy,p.nxCells);
      int j = p.cellHead[c];
      while(j!=-1){
        float dx = xi - p.x[j];
        float dy = yi - p.y[j];
        float r  = sqrtf(dx*dx + dy*dy);
        rhoi += W_wendland2D(r, p.h);  // masa=1 (MVP)
        j = p.next[j];
      }
    }
  }
  p.rho[i] = fmaxf(rhoi, 0.1f*p.rho0);
  p.P[i]   = p.cs*p.cs * (p.rho[i] - p.rho0);
}

// --- Fuerzas ---
__global__ void kForces(Particles p){
  int i=blockIdx.x*blockDim.x+threadIdx.x; if(i>=p.N) return;

  float xi=p.x[i], yi=p.y[i];
  float axi = p.gx, ayi = p.gy;

  int cx=int(floorf(xi/p.cellSize)); cx=clampi(cx,0,p.nxCells-1);
  int cy=int(floorf(yi/p.cellSize)); cy=clampi(cy,0,p.nyCells-1);

  #pragma unroll
  for(int oy=-1; oy<=1; ++oy){
    #pragma unroll
    for(int ox=-1; ox<=1; ++ox){
      int ncx = clampi(cx+ox,0,p.nxCells-1);
      int ncy = clampi(cy+oy,0,p.nyCells-1);
      int c   = cellId(ncx,ncy,p.nxCells);
      int j = p.cellHead[c];
      while(j!=-1){
        if(j!=i){
          float dx = xi - p.x[j];
          float dy = yi - p.y[j];
          float r2 = dx*dx + dy*dy;
          float r  = sqrtf(r2) + 1e-12f;
          if(r < p.h){
            float grad = dW_wendland2D(r, p.h);
            float pij  = (p.P[i]/(p.rho[i]*p.rho[i]) + p.P[j]/(p.rho[j]*p.rho[j]));
            float invr = 1.0f/r;
            axi += -pij * grad * dx * invr;
            ayi += -pij * grad * dy * invr;

            // Viscosidad simple
            float dvx = p.vx[j]-p.vx[i];
            float dvy = p.vy[j]-p.vy[i];
            axi += p.nu * dvx;
            ayi += p.nu * dvy;
          }
        }
        j = p.next[j];
      }
    }
  }

  // Paredes no deslizantes simples
  float wall_d = 2.0f*p.dp;
  if(yi < wall_d || (p.Ly-yi) < wall_d){
    axi -= 5.0f*p.vx[i];
    ayi -= 5.0f*p.vy[i];
  }

  p.ax[i]=axi; p.ay[i]=ayi;
}

// --- Integración Verlet ---
__global__ void kVerlet_halfkick(Particles p){
  int i=blockIdx.x*blockDim.x+threadIdx.x; if(i>=p.N) return;
  p.vx[i] += 0.5f * p.dt * p.ax[i];
  p.vy[i] += 0.5f * p.dt * p.ay[i];
}
__global__ void kVerlet_drift(Particles p){
  int i=blockIdx.x*blockDim.x+threadIdx.x; if(i>=p.N) return;
  p.x[i]  += p.dt * p.vx[i];
  p.y[i]  += p.dt * p.vy[i];

  if(p.x[i] < 0.0f)  p.x[i] += p.Lx;
  if(p.x[i] >= p.Lx) p.x[i] -= p.Lx;
  if(p.y[i] < 0.0f){ p.y[i]=0.0f;  p.vy[i]=0.0f; }
  if(p.y[i] >  p.Ly){ p.y[i]=p.Ly;  p.vy[i]=0.0f; }
}
__global__ void kVerlet_halfkick_end(Particles p){
  int i=blockIdx.x*blockDim.x+threadIdx.x; if(i>=p.N) return;
  p.vx[i] += 0.5f * p.dt * p.ax[i];
  p.vy[i] += 0.5f * p.dt * p.ay[i];
}

// --- Perfil u(y) ---
__global__ void kProfileUy(const Particles p, float *acc_u, int *acc_c){
#if WRITE_PROFILE
  int i=blockIdx.x*blockDim.x+threadIdx.x; if(i>=p.N) return;
  int bin = int((p.y[i]/p.Ly) * YBINS);
  if(bin<0) bin=0; else if(bin>=YBINS) bin=YBINS-1;
  atomicAdd(&acc_u[bin], p.vx[i]);
  atomicAdd(&acc_c[bin], 1);
#endif
}

// Inicialización
void init_particles(std::vector<float>& x,std::vector<float>& y,
                    std::vector<float>& vx,std::vector<float>& vy,
                    int& N, const SimParams& sp)
{
  std::vector<float> px, py;
  for(float yy=sp.dp*1.0f; yy<=sp.Ly-sp.dp*1.0f; yy+=sp.dp){
    for(float xx=0.0f; xx<=sp.Lx-sp.dp; xx+=sp.dp){
      px.push_back(xx + 0.5f*sp.dp);
      py.push_back(yy);
    }
  }
  N = (int)px.size();
  x.resize(N); y.resize(N); vx.assign(N,0.0f); vy.assign(N,0.0f);
  for(int i=0;i<N;++i){ x[i]=px[i]; y[i]=py[i]; }
}

int main(){
  SimParams sp; sp.finalize();

  // Host
  int N=0; std::vector<float> hx,hy,hvx,hvy; init_particles(hx,hy,hvx,hvy,N,sp);
  size_t nBytes = size_t(N)*sizeof(float);

  // Device alloc
  Particles p{};
  p.N=N; p.nxCells=sp.nxCells; p.nyCells=sp.nyCells;
  p.cellSize=sp.cellSize; p.h=sp.h; p.rho0=sp.rho0; p.cs=sp.cs; p.nu=sp.nu;
  p.gx=sp.gx; p.gy=sp.gy; p.Lx=sp.Lx; p.Ly=sp.Ly; p.dt=sp.dt; p.dp=sp.dp;

  CUDA_CHECK(hipMalloc(&p.x, nBytes), "hipMalloc");
  CUDA_CHECK(hipMalloc(&p.y, nBytes), "hipMalloc");
  CUDA_CHECK(hipMalloc(&p.vx,nBytes), "hipMalloc");
  CUDA_CHECK(hipMalloc(&p.vy,nBytes), "hipMalloc");
  CUDA_CHECK(hipMalloc(&p.ax,nBytes), "hipMalloc");
  CUDA_CHECK(hipMalloc(&p.ay,nBytes), "hipMalloc");
  CUDA_CHECK(hipMalloc(&p.rho,nBytes), "hipMalloc");
  CUDA_CHECK(hipMalloc(&p.P,  nBytes), "hipMalloc");
  CUDA_CHECK(hipMalloc(&p.next, N*sizeof(int)), "hipMalloc");
  CUDA_CHECK(hipMalloc(&p.cellHead, sp.nxCells*sp.nyCells*sizeof(int)), "hipMalloc");

  CUDA_CHECK(hipMemcpy(p.x, hx.data(), nBytes, hipMemcpyHostToDevice), "hipMemcpy");
  CUDA_CHECK(hipMemcpy(p.y, hy.data(), nBytes, hipMemcpyHostToDevice), "hipMemcpy");
  CUDA_CHECK(hipMemcpy(p.vx,hvx.data(), nBytes, hipMemcpyHostToDevice), "hipMemcpy");
  CUDA_CHECK(hipMemcpy(p.vy,hvy.data(), nBytes, hipMemcpyHostToDevice), "hipMemcpy");

  dim3 block(BLOCK_SIZE);
  dim3 gridP((N+block.x-1)/block.x);
  dim3 gridHead((sp.nxCells*sp.nyCells + block.x -1)/block.x);

  // Buffers de perfil
  float *d_prof_u=nullptr; int *d_prof_c=nullptr;
#if WRITE_PROFILE
  CUDA_CHECK(hipMalloc(&d_prof_u, YBINS*sizeof(float)), "hipMalloc");
  CUDA_CHECK(hipMalloc(&d_prof_c, YBINS*sizeof(int)),   "hipMalloc");
  CUDA_CHECK(hipMemset(d_prof_u, 0, YBINS*sizeof(float)), "hipMemset");
  CUDA_CHECK(hipMemset(d_prof_c, 0, YBINS*sizeof(int)),   "hipMemset");
#endif

  // Bucle de tiempo
  GpuTimer Tall; Tall.tic();
  for(int s=0; s<sp.steps; ++s){
    kVerlet_halfkick<<<gridP, block>>>(p);

    kResetHeads<<<gridHead, block>>>(p.cellHead, sp.nxCells*sp.nyCells);
    kBuildLinked<<<gridP, block>>>(p.x,p.y,p.next,p.cellHead,p.N, p.cellSize, p.nxCells, p.nyCells);
    kDensityPressure<<<gridP, block>>>(p);
    kForces<<<gridP, block>>>(p);

    kVerlet_drift<<<gridP, block>>>(p);

    kResetHeads<<<gridHead, block>>>(p.cellHead, sp.nxCells*sp.nyCells);
    kBuildLinked<<<gridP, block>>>(p.x,p.y,p.next,p.cellHead,p.N, p.cellSize, p.nxCells, p.nyCells);
    kDensityPressure<<<gridP, block>>>(p);
    kForces<<<gridP, block>>>(p);

    kVerlet_halfkick_end<<<gridP, block>>>(p);
  }
  hipDeviceSynchronize();
  float ms_total = Tall.toc();

  double updates = double(N) * double(sp.steps);
  double mpups   = updates / (ms_total*1e3);
  std::printf("N=%d steps=%d  Time=%.1f ms  MPUPS=%.2f  (block=%d)\n",
              N, sp.steps, ms_total, mpups, BLOCK_SIZE);

#if WRITE_PROFILE
  kProfileUy<<<gridP, block>>>(p, d_prof_u, d_prof_c);
  std::vector<float> hpu(YBINS,0.0f); std::vector<int> hpc(YBINS,0);
  CUDA_CHECK(hipMemcpy(hpu.data(), d_prof_u, YBINS*sizeof(float), hipMemcpyDeviceToHost), "hipMemcpy");
  CUDA_CHECK(hipMemcpy(hpc.data(), d_prof_c, YBINS*sizeof(int),   hipMemcpyDeviceToHost), "hipMemcpy");

  FILE* f = std::fopen("profile_uy_cuda.csv","w");
  if(f){
    std::fprintf(f,"bin,y,u_mean\n");
    for(int b=0;b<YBINS;++b){
      float y = ( (b+0.5f)/float(YBINS) ) * sp.Ly;
      float u = (hpc[b]>0) ? (hpu[b]/float(hpc[b])) : 0.0f;
      std::fprintf(f,"%d,%.7f,%.7f\n", b, y, u);
    }
    std::fclose(f);
    std::printf("Perfil u(y) guardado: profile_uy_cuda.csv (YBINS=%d)\n", YBINS);
  }
  cudaFreeChecked(d_prof_u);
  cudaFreeChecked(d_prof_c);
#endif

  cudaFreeChecked(p.x); cudaFreeChecked(p.y);
  cudaFreeChecked(p.vx); cudaFreeChecked(p.vy);
  cudaFreeChecked(p.ax); cudaFreeChecked(p.ay);
  cudaFreeChecked(p.rho); cudaFreeChecked(p.P);
  cudaFreeChecked(p.next); cudaFreeChecked(p.cellHead);
  return 0;
}
