#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_math_constants.h> // mantenemos las constantes CUDA
#include <cfloat>           // FLT_EPSILON (CUDA 8 no tiene CUDART_EPSILON)
#include "dam_kernels.hpp"

namespace dsph { namespace dambreak {

__global__ void k_taitPressure(std::uint32_t n, float rho0, float cs, float gamma,
                               const float* __restrict__ rho,
                               float* __restrict__ p) {
  const std::uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) return;
  // Evita dividir por cero: CUDA 8.0 no define CUDART_EPSILON; usa FLT_EPSILON
  const float rr = fmaxf(rho[i], FLT_EPSILON) / rho0;
  const float term = powf(rr, gamma) - 1.0f;
  // p = (rho0 * cs^2 / gamma) * ( (rho/rho0)^gamma - 1 )
  p[i] = (rho0 * cs * cs / gamma) * term;
}

__global__ void k_addGravity(std::uint32_t n, float dt, float gx, float gy, float gz,
                             float* __restrict__ vx,
                             float* __restrict__ vy,
                             float* __restrict__ vz) {
  const std::uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) return;
  vx[i] += dt * gx;
  vy[i] += dt * gy;
  vz[i] += dt * gz;
}

__global__ void k_integrateEuler(std::uint32_t n, float dt,
                                 float* __restrict__ x,
                                 float* __restrict__ y,
                                 float* __restrict__ z,
                                 const float* __restrict__ vx,
                                 const float* __restrict__ vy,
                                 const float* __restrict__ vz) {
  const std::uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) return;
  x[i] += dt * vx[i];
  y[i] += dt * vy[i];
  z[i] += dt * vz[i];
}

static inline dim3 blocks(std::uint32_t n, std::uint32_t bs) {
  return dim3((n + bs - 1u) / bs);
}

void taitPressure(const Domain& dom,
                  const float* __restrict__ rho,
                  float* __restrict__ p,
                  hipStream_t stream) {
  const std::uint32_t bs = 256u;
  k_taitPressure<<<blocks(dom.n, bs), dim3(bs), 0, stream>>>(
      dom.n, dom.rho0, dom.cs, dom.gamma, rho, p);
}

void addGravity(const Domain& dom,
                float* __restrict__ vx,
                float* __restrict__ vy,
                float* __restrict__ vz,
                hipStream_t stream) {
  const std::uint32_t bs = 256u;
  k_addGravity<<<blocks(dom.n, bs), dim3(bs), 0, stream>>>(
      dom.n, dom.dt, dom.gx, dom.gy, dom.gz, vx, vy, vz);
}

void integrateEuler(const Domain& dom,
                    float* __restrict__ x,
                    float* __restrict__ y,
                    float* __restrict__ z,
                    const float* __restrict__ vx,
                    const float* __restrict__ vy,
                    const float* __restrict__ vz,
                    hipStream_t stream) {
  const std::uint32_t bs = 256u;
  k_integrateEuler<<<blocks(dom.n, bs), dim3(bs), 0, stream>>>(
      dom.n, dom.dt, x, y, z, vx, vy, vz);
}

void runStep(const Domain& dom,
             const float* __restrict__ rho,
             float* __restrict__ p,
             float* __restrict__ x,
             float* __restrict__ y,
             float* __restrict__ z,
             float* __restrict__ vx,
             float* __restrict__ vy,
             float* __restrict__ vz,
             hipStream_t stream) {
  taitPressure(dom, rho, p, stream);
  addGravity(dom, vx, vy, vz, stream);
  integrateEuler(dom, x, y, z, vx, vy, vz, stream);
  hipGetLastError();
}

}} // namespace dsph::dambreak


